
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <chrono>

#define RAIO 10
#define TAMANHO 20

__constant__ float const_stencilWeight[10000];

// base case
__global__ void stencil(float *src, float *dst, int size, int raio, float *stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += raio+1;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -raio;i < raio; i++)
    {
        out += src[idx+i] * stencilWeight[i+raio];
    }
    dst[idx] = out;
}

// read only cache stencil coefficients
__global__ void stencilReadOnly1(float *src, float *dst, int size, int raio, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += raio+1;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -raio;i < raio; i++){
        out += src[idx+i] * stencilWeight[i+raio];
    }
    dst[idx] = out;
}

// read only data
__global__ void stencilReadOnly2(float *src, float *dst, int size, int raio, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += raio+1;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -raio;i < raio; i++)
    {
        out += src[idx+i] * stencilWeight[i+raio];
    }
    dst[idx] = out;
}

// read only coefficients and data
__global__ void stencilReadOnly3(float *src, float *dst, int size, int raio, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += raio+1;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -raio;i < raio; i++)
    {
        out += src[idx+i] * stencilWeight[i+raio];
    }
    dst[idx] = out;
}

// constat memory coefficients
__global__ void stencilConst1(float *src, float *dst, int size, int raio)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += raio+1;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -raio;i < raio; i++)
    {
        out += src[idx+i] * const_stencilWeight[i+raio];
    }
    dst[idx] = out;
}

// constant memory coefficients and data through read only cache
__global__ void stencilConst2(float *src, float *dst, int size, int raio)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += raio+1;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -raio;i < raio; i++)
    {
        out += src[idx+i] * const_stencilWeight[i+raio];
    }
    dst[idx] = out;
}

// constant memory coefficients and data from shared 
__global__ void stencilShared1(float *src, float *dst, int size, int raio)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float buffer[1024+11];
    for(int i = threadIdx.x; i < 1024+21; i = i + 1024)
    {
        buffer[i] = src[idx+i];
    }
    idx += raio+1;
    if (idx >= size)
        return;
    
    __syncthreads();
    float out = 0;
    #pragma unroll
    for(int i = -raio;i < raio; i++)
    {
        out += buffer[threadIdx.x+raio+i] * const_stencilWeight[i+raio];
    }
    dst[idx] = out;
}

// constant memory coefficients and data from shared thorugh read only
__global__ void stencilShared2(float *src, float *dst, int size, int raio)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float buffer[1024+11];
    for(int i = threadIdx.x; i < 1024+21; i = i + 1024)
    {
        buffer[i] = src[idx+i];
    }
    idx += raio;
    if (idx >= size)
        return;
    
    __syncthreads();
    float out = 0;
    #pragma unroll
    for(int i = -raio;i < raio; i++)
    {
        out += buffer[threadIdx.x+raio+i] * const_stencilWeight[i+raio];
    }
    dst[idx] = out;
}

bool verify(float *arr, float *corr, int count, int raio){
    // skip the first elements since they may be wrong
    for(int i = raio+1; i < count; i++){
        if(arr[i] != corr[i]){   
            std::cout << "check failed" << i << " " << arr[i] << " != " << corr[i] << std::endl;
            exit(1);
        }
    }
    return true;
}

int main()
{

    int tamanho = 1 << TAMANHO; 

    float *a;
    float *b;
    float *bOut;
    float *bCorr;
    float *weights;
    hipMalloc(&a, sizeof(float)*tamanho);
    hipMalloc(&b, sizeof(float)*tamanho);
    hipHostMalloc(&bOut, sizeof(float)*tamanho, hipHostMallocDefault);
    hipMallocManaged(&bCorr, sizeof(float)*tamanho);
    hipMallocManaged(&weights, sizeof(float)*(2*RAIO+1));

    hipDeviceSynchronize();

    for(int i = 0; i < tamanho;i++){
        //a[i] = 0;
        //b[i] = 0;
        bCorr[i] = 0;
    }

    hipMemset(a, 1, tamanho);
    hipMemset(b, 1, tamanho);
    hipMemset(bCorr, 1, tamanho);
    hipMemset(bOut, 1, tamanho);

    hipDeviceSynchronize();    
    
    int blockSize = 1024;
    int blocks = 10000;
    for(int i = 0; i < 2*RAIO+1; ++i)
        weights[i] = i-10;
    
    
    hipDeviceSynchronize();    

    // copy to constant memory    
    hipMemcpyToSymbol(HIP_SYMBOL(const_stencilWeight), weights, sizeof(float)*(2*RAIO+1));
    //cudaMemcpy(const_stencilWeight, weights, sizeof(float)*(2*RAIO+1), cudaMemcpyDefault);

    // run the basic case once to get the "correct" results
    stencil<<<blocks, blockSize>>>(a, bCorr, tamanho, RAIO, weights);
    hipDeviceSynchronize();    

    stencil<<<blocks, blockSize>>>(a, b, tamanho, RAIO, weights);
    hipDeviceSynchronize(); 
    hipMemcpy(bOut, b, sizeof(float)*tamanho, hipMemcpyDefault);
    verify(bOut, bCorr, 1000, RAIO);

    hipSetDevice(0); 

    float minTime = 10000;
    for(int i  = 0; i < 10; i++){
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        stencil<<<blocks, blockSize>>>(a, b, tamanho, RAIO, weights);
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();
        
        hipMemcpy(bOut, b, sizeof(float)*tamanho, hipMemcpyDefault);
        verify(bOut, bCorr, 1000, RAIO);  

	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "Non optimized " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;
    std::cout << std::endl;

    for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        stencilReadOnly1<<<blocks, blockSize>>>(a, b, tamanho, RAIO, weights);
        hipDeviceSynchronize();  
        end = std::chrono::system_clock::now();
        
        hipMemcpy(bOut, b, sizeof(float)*tamanho, hipMemcpyDefault);
        verify(bOut, bCorr, 1000, RAIO);  
        
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "read only cache stencil coefficients " <<(blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;
    for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        stencilReadOnly2<<<blocks, blockSize>>>(a, b, tamanho, RAIO, weights);
        hipDeviceSynchronize();  
        end = std::chrono::system_clock::now();
        
        (hipMemcpy(bOut, b, sizeof(float)*tamanho, hipMemcpyDefault));
        verify(bOut, bCorr, 1000, RAIO);  
        
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "read only data " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;
    for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        stencilReadOnly3<<<blocks, blockSize>>>(a, b, tamanho, RAIO, weights);
        hipDeviceSynchronize();  
        end = std::chrono::system_clock::now();
        
        hipMemcpy(bOut, b, sizeof(float)*tamanho, hipMemcpyDefault);
        verify(bOut, bCorr, 1000, RAIO);  
        
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "read only coefficients and data " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;

    std::cout << std::endl;

        for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        stencilConst1<<<blocks, blockSize>>>(a, b, tamanho, RAIO);
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();

        (hipMemcpy(bOut, b, sizeof(float)*tamanho, hipMemcpyDefault));
        verify(bOut, bCorr, 1000, RAIO);  
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients " << (blockSize*blocks)/minTime << " updates/s" << std::endl;

    minTime = 10000;

    for(int i  = 0; i < 10; i++){
        hipDeviceSynchronize();  
        
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        stencilConst2<<<blocks, blockSize>>>(a, b, tamanho, RAIO);
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();

        (hipMemcpy(bOut, b, sizeof(float)*tamanho, hipMemcpyDefault));
        verify(bOut, bCorr, 1000, RAIO);  
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients and data through read only cache " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    std::cout << std::endl;


    minTime = 10000;
            for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        stencilShared1<<<blocks, blockSize>>>(a, b, tamanho, RAIO);
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();
        
        (hipMemcpy(bOut, b, sizeof(float)*tamanho, hipMemcpyDefault));
        verify(bOut, bCorr, 1000, RAIO);  
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients and data from shared " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;
    minTime = 10000;
    for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        stencilShared2<<<blocks, blockSize>>>(a, b, tamanho, RAIO);
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();
        
        hipMemcpy(bOut, b, sizeof(float)*tamanho, hipMemcpyDefault);
        verify(bOut, bCorr, 1000, RAIO);  
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients and data from shared thorugh read only " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;

}
