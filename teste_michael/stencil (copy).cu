
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <chrono>

__constant__ float const_stencilWeight[21];

// base case
__global__ void stencil(float *src, float *dst, int size, float *stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += src[idx+i] * stencilWeight[i+10];
    }
    dst[idx] = out;
}

// read only cache stencil coefficients
__global__ void stencilReadOnly1(float *src, float *dst, int size, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += src[idx+i] * stencilWeight[i+10];
    }
    dst[idx] = out;
}

// read only data
__global__ void stencilReadOnly2(float *src, float *dst, int size, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += src[idx+i] * stencilWeight[i+10];
    }
    dst[idx] = out;
}

// read only coefficients and data
__global__ void stencilReadOnly3(float *src, float *dst, int size, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += src[idx+i] * stencilWeight[i+10];
    }
    dst[idx] = out;
}

// constat memory coefficients
__global__ void stencilConst1(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += src[idx+i] * const_stencilWeight[i+10];
    }
    dst[idx] = out;
}

// constant memory coefficients and data through read only cache
__global__ void stencilConst2(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += src[idx+i] * const_stencilWeight[i+10];
    }
    dst[idx] = out;
}

// constant memory coefficients and data from shared 
__global__ void stencilShared1(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float buffer[1024+21];
    for(int i = threadIdx.x; i < 1024+21; i = i + 1024)
    {
        buffer[i] = src[idx+i];
    }
    idx += 11;
    if (idx >= size)
        return;
    
    __syncthreads();
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += buffer[threadIdx.x+10+i] * const_stencilWeight[i+10];
    }
    dst[idx] = out;
}

// constant memory coefficients and data from shared thorugh read only
__global__ void stencilShared2(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float buffer[1024+21];
    for(int i = threadIdx.x; i < 1024+21; i = i + 1024)
    {
        buffer[i] = src[idx+i];
    }
    idx += 11;
    if (idx >= size)
        return;
    
    __syncthreads();
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += buffer[threadIdx.x+10+i] * const_stencilWeight[i+10];
    }
    dst[idx] = out;
}

bool verify(float *arr, float *corr, int count)
{
    // skip the first elements since they may be wrong
    for(int i = 11; i < count; i++)
    {
        if(arr[i] != corr[i])
        {   
            std::cout << "check failed" << i << " " << arr[i] << " != " << corr[i] << std::endl;
            exit(1);
        }
    }
    return true;
}

int main()
{
    float *a;
    float *b;
    float *bOut;
    float *bCorr;
    float *weights;
    (hipMalloc(&a, sizeof(float)*102400000));
    (hipMalloc(&b, sizeof(float)*102400000));
    (hipHostMalloc(&bOut, sizeof(float)*102400000, hipHostMallocDefault));
    (hipMallocManaged(&bCorr, sizeof(float)*102400000));
    (hipMallocManaged(&weights, sizeof(float)*21));

    hipDeviceSynchronize();    

    for(int i = 0; i < 102400000;i++)
    {
        //a[i] = 0;
        //b[i] = 0;
        bCorr[i] = 0;
    }

    hipMemset(a, 1, 102400000);
    hipMemset(b, 1, 102400000);
    hipMemset(bCorr, 1, 102400000);
    hipMemset(bOut, 1, 102400000);

    hipDeviceSynchronize();    
    
    int blockSize = 1024;
    int blocks = 10000;
    for(int i = 0; i < 21;i++)
        weights[i] = i-10;
    
    
    hipDeviceSynchronize();    


    // copy to constant memory    
    hipMemcpyToSymbol(HIP_SYMBOL(const_stencilWeight), weights, sizeof(float)*21);

    // run the basic case once to get the "correct" results
    ((stencil<<<blocks, blockSize>>>(a, bCorr, 10240000, weights)));
    hipDeviceSynchronize();    

    ((stencil<<<blocks, blockSize>>>(a, b, 10240000, weights)));
    hipDeviceSynchronize(); 
    (hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
    verify(bOut, bCorr, 1000);

    hipSetDevice(0); 


    float minTime = 10000;
    for(int i  = 0; i < 10; i++)
    {
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        ((stencil<<<blocks, blockSize>>>(a, b, 10240000, weights)));
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();
        
        (hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  

	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "Non optimized " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;
    std::cout << std::endl;

    for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        ((stencilReadOnly1<<<blocks, blockSize>>>(a, b, 10240000, weights)));
        hipDeviceSynchronize();  
        end = std::chrono::system_clock::now();
        
        (hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
        
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "read only cache stencil coefficients " <<(blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;
    for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        ((stencilReadOnly2<<<blocks, blockSize>>>(a, b, 10240000, weights)));
        hipDeviceSynchronize();  
        end = std::chrono::system_clock::now();
        
        (hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
        
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "read only data " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;
    for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        ((stencilReadOnly3<<<blocks, blockSize>>>(a, b, 10240000, weights)));
        hipDeviceSynchronize();  
        end = std::chrono::system_clock::now();
        
        (hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
        
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "read only coefficients and data " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;

    std::cout << std::endl;

        for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        ((stencilConst1<<<blocks, blockSize>>>(a, b, 10240000)));
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();

        (hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients " << (blockSize*blocks)/minTime << " updates/s" << std::endl;

    minTime = 10000;


        for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        ((stencilConst2<<<blocks, blockSize>>>(a, b, 10240000)));
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();

        (hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients and data through read only cache " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    std::cout << std::endl;


    minTime = 10000;
            for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        ((stencilShared1<<<blocks, blockSize>>>(a, b, 10240000)));
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();
        
        (hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients and data from shared " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;
    minTime = 10000;
    for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        ((stencilShared2<<<blocks, blockSize>>>(a, b, 10240000)));
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();
        
        (hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients and data from shared thorugh read only " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;


}
