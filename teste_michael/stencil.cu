#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <chrono>

#define DEBUG_DEV

#ifdef DEBUG_DEV
#define getErrorCuda(command)\
		command;\
		hipDeviceSynchronize();\
		hipDeviceSynchronize();\
		if (hipPeekAtLastError() != hipSuccess){\
			std::cout << #command << " : " << hipGetErrorString(hipGetLastError())\
			 << " in file " << __FILE__ << " at line " << __LINE__ << std::endl;\
			exit(1);\
		}
#endif
#ifndef DEBUG_DEV
#define getErrorCuda(command) command;
#endif

__constant__ float const_stencilWeight[21];


// base case
__global__ void stencil(float *src, float *dst, int size, float *stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += src[idx+i] * stencilWeight[i+10];
    }
    dst[idx] = out;
}

// read only cache stencil coefficients
__global__ void stencilReadOnly1(float *src, float *dst, int size, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += src[idx+i] * __ldg(&stencilWeight[i+10]);
    }
    dst[idx] = out;
}

// read only data
__global__ void stencilReadOnly2(float *src, float *dst, int size, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += __ldg(&src[idx+i]) * stencilWeight[i+10];
    }
    dst[idx] = out;
}

// read only coefficients and data
__global__ void stencilReadOnly3(float *src, float *dst, int size, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += __ldg(&src[idx+i]) * __ldg(&stencilWeight[i+10]);
    }
    dst[idx] = out;
}

// constat memory coefficients
__global__ void stencilConst1(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += src[idx+i] * const_stencilWeight[i+10];
    }
    dst[idx] = out;
}

// constant memory coefficients and data through read only cache
__global__ void stencilConst2(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += __ldg(&src[idx+i]) * const_stencilWeight[i+10];
    }
    dst[idx] = out;
}

// constant memory coefficients and data from shared 
__global__ void stencilShared1(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float buffer[1024+21];
    for(int i = threadIdx.x; i < 1024+21; i = i + 1024)
    {
        buffer[i] = src[idx+i];
    }
    idx += 11;
    if (idx >= size)
        return;
    
    __syncthreads();
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += buffer[threadIdx.x+10+i] * const_stencilWeight[i+10];
    }
    dst[idx] = out;
}

// constant memory coefficients and data from shared thorugh read only
__global__ void stencilShared2(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float buffer[1024+21];
    for(int i = threadIdx.x; i < 1024+21; i = i + 1024)
    {
        buffer[i] = __ldg(&src[idx+i]);
    }
    idx += 11;
    if (idx >= size)
        return;
    
    __syncthreads();
    float out = 0;
    #pragma unroll
    for(int i = -10;i < 10; i++)
    {
        out += buffer[threadIdx.x+10+i] * const_stencilWeight[i+10];
    }
    dst[idx] = out;
}

bool verify(float *arr, float *corr, int count)
{
    // skip the first elements since they may be wrong
    for(int i = 11; i < count; i++)
    {
        if(arr[i] != corr[i])
        {   
            std::cout << "check failed" << i << " " << arr[i] << " != " << corr[i] << std::endl;
            exit(1);
        }
    }
}

int main()
{
    float *a;
    float *b;
    float *bOut;
    float *bCorr;
    float *weights;
    getErrorCuda(hipMalloc(&a, sizeof(float)*102400000));
    getErrorCuda(hipMalloc(&b, sizeof(float)*102400000));
    getErrorCuda(hipHostMalloc(&bOut, sizeof(float)*102400000));
    getErrorCuda(hipMallocManaged(&bCorr, sizeof(float)*102400000));
    getErrorCuda(hipMallocManaged(&weights, sizeof(float)*21));

    hipDeviceSynchronize();    

    for(int i = 0; i < 102400000;i++)
    {
        //a[i] = 0;
        //b[i] = 0;
        bCorr[i] = 0;
    }

    hipMemset(a, 1, 102400000);
    hipMemset(b, 1, 102400000);
    hipMemset(bCorr, 1, 102400000);
    hipMemset(bOut, 1, 102400000);

    hipDeviceSynchronize();    
    
    int blockSize = 1024;
    int blocks = 10000;
    for(int i = 0; i < 21;i++)
        weights[i] = i-10;
    
    
    hipDeviceSynchronize();    


    // copy to constant memory    
    hipMemcpyToSymbol(HIP_SYMBOL(const_stencilWeight), weights, sizeof(float)*21);

    // run the basic case once to get the "correct" results
    getErrorCuda((stencil<<<blocks, blockSize>>>(a, bCorr, 10240000, weights)));
    hipDeviceSynchronize();    

    getErrorCuda((stencil<<<blocks, blockSize>>>(a, b, 10240000, weights)));
    hipDeviceSynchronize(); 
    getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
    verify(bOut, bCorr, 1000);

    hipSetDevice(0); 


    float minTime = 10000;
    for(int i  = 0; i < 10; i++)
    {
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        getErrorCuda((stencil<<<blocks, blockSize>>>(a, b, 10240000, weights)));
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();
        
        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  

	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "Non optimized " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;
    std::cout << std::endl;

    for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        getErrorCuda((stencilReadOnly1<<<blocks, blockSize>>>(a, b, 10240000, weights)));
        hipDeviceSynchronize();  
        end = std::chrono::system_clock::now();
        
        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
        
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "read only cache stencil coefficients " <<(blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;
    for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        getErrorCuda((stencilReadOnly2<<<blocks, blockSize>>>(a, b, 10240000, weights)));
        hipDeviceSynchronize();  
        end = std::chrono::system_clock::now();
        
        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
        
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "read only data " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;
    for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        getErrorCuda((stencilReadOnly3<<<blocks, blockSize>>>(a, b, 10240000, weights)));
        hipDeviceSynchronize();  
        end = std::chrono::system_clock::now();
        
        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
        
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "read only coefficients and data " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;

    std::cout << std::endl;

        for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        getErrorCuda((stencilConst1<<<blocks, blockSize>>>(a, b, 10240000)));
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();

        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients " << (blockSize*blocks)/minTime << " updates/s" << std::endl;

    minTime = 10000;


        for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        getErrorCuda((stencilConst2<<<blocks, blockSize>>>(a, b, 10240000)));
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();

        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients and data through read only cache " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    std::cout << std::endl;


    minTime = 10000;
            for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        getErrorCuda((stencilShared1<<<blocks, blockSize>>>(a, b, 10240000)));
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();
        
        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients and data from shared " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;
    minTime = 10000;
    for(int i  = 0; i < 10; i++)
    {
        hipDeviceSynchronize();  
        
        std::chrono::time_point<std::chrono::system_clock> start, end;
	    start = std::chrono::system_clock::now();
        getErrorCuda((stencilShared2<<<blocks, blockSize>>>(a, b, 10240000)));
        hipDeviceSynchronize();    
        end = std::chrono::system_clock::now();
        
        getErrorCuda(hipMemcpy(bOut, b, sizeof(float)*10240000, hipMemcpyDefault));
        verify(bOut, bCorr, 1000);  
	    std::chrono::duration<float> elapsed_seconds = end-start;
        minTime = std::min(elapsed_seconds.count(), minTime);
    }
    std::cout << "constant memory coefficients and data from shared thorugh read only " << (blockSize*blocks)/minTime << " updates/s" << std::endl;
    minTime = 10000;


}
