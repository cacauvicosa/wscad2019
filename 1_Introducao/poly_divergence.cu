#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This example demonstrates a simple vector sum on the GPU and on the host.
 * sumArraysOnGPU splits the work of the vector sum across CUDA threads on the
 * GPU. Only a single thread block is used in this small case, for simplicity.
 * sumArraysOnHost sequentially iterates through vector elements on the host.
 * This version of sumArrays adds host timers to measure GPU and CPU
 * performance.
 */


int main() {
    int nElem = 1 << POWER;
    float elapsed_time;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);


    size_t nBytes = nElem * sizeof(float);

    float* h_polinomy = (float*)malloc(nBytes);

    float* d_polinomy;
    float* d_results;
    hipMalloc((float**)&d_polinomy, nBytes);
    hipMalloc((float**)&d_results, nBytes);
  
    int iLen = 512;
    dim3 block (iLen);
    dim3 grid  ((nElem + block.x - 1) / block.x);


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event
    hipEventRecord(start, 0); 
    poli_without_divergence<<<grid, block>>>(d_polinomy, nElem);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // calculate elapsed time
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Without divergence - Measured time for parallel execution = %.6fms\n",
           elapsed_time );
 
    hipMemcpy(h_polinomy, d_polinomy, nBytes, hipMemcpyHostToDevice);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event
    hipEventRecord(start, 0); 
    poli_div<<<grid, block>>>(d_polinomy, nElem);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // calculate elapsed time
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("With Divergence - Measured time for parallel execution = %.6fms\n",
           elapsed_time );

    hipMemcpy(h_polinomy, d_polinomy, nBytes, hipMemcpyHostToDevice);
    
    
    hipFree(d_polinomy);
    free(h_polinomy);
}
