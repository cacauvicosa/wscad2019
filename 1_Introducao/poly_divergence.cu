#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This example demonstrates a simple vector sum on the GPU and on the host.
 * sumArraysOnGPU splits the work of the vector sum across CUDA threads on the
 * GPU. Only a single thread block is used in this small case, for simplicity.
 * sumArraysOnHost sequentially iterates through vector elements on the host.
 * This version of sumArrays adds host timers to measure GPU and CPU
 * performance.
 */


__global__ void poli_warp(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    float x; 
    if (idx < N) {
       x = poli[idx];
       poli[idx] = 5 + x * ( 7 - x * (9 + x * (5 + x * (5 + x))))- 1.0f/x + 3.0f/(x*x) + x/5.0f;                 
    }
    poli[idx] = x;
}

int main() {
    int nElem = 1 << POWER;
    float elapsed_time;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);


    size_t nBytes = nElem * sizeof(float);

    float* h_polinomy = (float*)malloc(nBytes);

    float* d_polinomy;
    float* d_results;
    hipMalloc((float**)&d_polinomy, nBytes);
    hipMalloc((float**)&d_results, nBytes);
  
    int iLen = 512;
    dim3 block (iLen);
    dim3 grid  ((nElem + block.x - 1) / block.x);
    hipMemcpy(h_polinomy, d_polinomy, nBytes, hipMemcpyHostToDevice);
    poli_warp<<<grid, block>>>(d_polinomy, nElem);
    poli_warp<<<grid, block>>>(d_polinomy, nElem);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event
    hipEventRecord(start, 0); 
    poli_without_divergence<<<grid, block>>>(d_polinomy, nElem);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // calculate elapsed time
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Without divergence - Measured time for parallel execution = %.6fms\n",
           elapsed_time );
 
    hipMemcpy(h_polinomy, d_polinomy, nBytes, hipMemcpyHostToDevice);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event
    hipEventRecord(start, 0); 
    poli_div<<<grid, block>>>(d_polinomy, nElem);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // calculate elapsed time
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("With Divergence - Measured time for parallel execution = %.6fms\n",
           elapsed_time );

    hipMemcpy(d_polinomy, h_polinomy, nBytes, hipMemcpyDeviceToHost);
    
    
    hipFree(d_polinomy);
    free(h_polinomy);
}
