
#include <hip/hip_runtime.h>
#include <cstdio>

void initialize(float* polynomial, const size_t N) {
	for (size_t i = 0; i < N; ++i)
		polynomial[i] = static_cast<float>(i);
}

__global__ void kernel(float* polynomial, const size_t N) {
	int thread = blockIdx.x * blockDim.x + threadIdx.x;

	if (thread < N) {
		float x = polynomial[thread];

		polynomial[thread] = 3 * x * x - 7 * x + 5;
	}
}

int main(int argc, char** argv) {
	const size_t BLOCK_DIM = 128;
	const size_t N_STREAMS = 4;

	// Number of elements in the arrays
	size_t n_elem = 1u << 27u;
	size_t n_bytes = n_elem * sizeof(float);

	// Allocating the array in pinned host memory for async memcpy
	float* h_polynomial;
	hipHostAlloc(&h_polynomial, n_bytes, hipHostMallocDefault);

	// Initializing data on host
	initialize(h_polynomial, n_elem);

	// Allocating the device array
	float* d_polynomial;
	hipMalloc(&d_polynomial, n_bytes);

	// Number of elements per stream
	size_t n_elem_per_stream = n_elem / N_STREAMS;
	size_t n_bytes_per_stream = n_elem_per_stream * sizeof(float);

	// Events for time recording
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Grid and block
	dim3 block(BLOCK_DIM);
	dim3 grid((n_elem_per_stream + block.x - 1) / block.x);

	// Creating streams
	hipStream_t* streams = new hipStream_t[N_STREAMS];

	for (size_t i = 0; i < N_STREAMS; ++i)
		hipStreamCreate(&streams[i]);

	hipEventRecord(start, 0);

	//------------------------------------------------------- Asynchronous work
	for (size_t i = 0; i < N_STREAMS; ++i) {
		size_t offset = i * n_elem_per_stream;

		hipMemcpyAsync(&d_polynomial[offset], &h_polynomial[offset],
			n_bytes_per_stream, hipMemcpyHostToDevice, streams[i]);

		kernel<<<grid, block>>>(d_polynomial + offset, n_elem_per_stream);

		hipMemcpyAsync(&h_polynomial[offset], &d_polynomial[offset],
			n_bytes_per_stream, hipMemcpyDeviceToHost, streams[i]);
	}
	//-------------------------------------------------------------------------

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float time;

	hipEventElapsedTime(&time, start, stop);

	printf("Elapsed time: %.5f\n", time);

	// Destroying events
	hipEventDestroy(stop);
	hipEventDestroy(start);

	// Freeing memory
	delete[] streams;
	hipFree(d_polynomial);
	hipHostFree(h_polynomial);
}
