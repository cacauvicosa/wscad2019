#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * 
 */

__global__ void poly_div1(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
          
    if (idx < N) {
        float x = poli[idx];
        poli[idx] = 5 + x * ( 7 - x * (9 + x * (5 + x * (5 + x))))+x/5.0;
    }
}

__global__ void poly_div2(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
          
    if (idx < N) {
        float x = poli[idx];
        poli[idx] = 5 + x * ( 7 - x * (9 + x * (5 + x * (5 + x))))+x*0.2;
    }
}

__global__ void poly_div3(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
          
    if (idx < N) {
        float x = poli[idx];
        poli[idx] = 5 + x * ( 7 - x * (9 + x * (5 + x * (5 + x))))+5.0/x;
    }
}

__global__ void poly_div4(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
          
    if (idx < N) {
        float x = poli[idx];
        float y = 5.0/x;
        poli[idx] = 5 + x * ( 7 - x * (9 + x * (5 + x * (5 + x))))+y;
    }
}

__global__ void poly_div5(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
          
    if (idx < N) {
        float x = poli[idx];
        poli[idx] = 5 + x * ( 7 - x * (9 + x * (5 + x * (5 + x))))+1.0/x;
    }
}

__global__ void poly_div6(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
          
    if (idx < N) {
        float x = poli[idx];
        float y = 1.0/x;
        poli[idx] = 5 + x * ( 7 - x * (9 + x * (5 + x * (5 + x))))+y;
    }
}


__global__ void poly_div7(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
          
    if (idx < N) {
        float x = poli[idx];
        poli[idx] = 5 + x * ( 7 - x * (9 + x * (5 + x * (5 + x))))+1.0f/x;
    }
}

__global__ void poly_div8(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
          
    if (idx < N) {
        float x = poli[idx];
        poli[idx] = 5 + x * ( 7 - x * (9 + x * (5 + x * (5 + x))))+5.0f/x;
    }
}

int main() {
    int nElem = 1 << 27;
    float elapsed_time;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);


    size_t nBytes = nElem * sizeof(float);

    float* h_polinomy = (float*)malloc(nBytes);

    float* d_polinomy;
    float* d_results;
    hipMalloc((float**)&d_polinomy, nBytes);
    hipMalloc((float**)&d_results, nBytes);
  
    int iLen = 512;
    dim3 block (iLen);
    dim3 grid  ((nElem + block.x - 1) / block.x);


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event
    hipEventRecord(start, 0); 
    poly_div1<<<grid, block>>>(d_polinomy, nElem);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // calculate elapsed time
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Measured time for parallel execution = %.6fms\n",
           elapsed_time );

    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event
    hipEventRecord(start, 0); 
    poly_div2<<<grid, block>>>(d_polinomy, nElem);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // calculate elapsed time
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Measured time for parallel execution = %.6fms\n",
           elapsed_time );
    
hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event
    hipEventRecord(start, 0); 
    poly_div3<<<grid, block>>>(d_polinomy, nElem);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // calculate elapsed time
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Measured time for parallel execution = %.6fms\n",
           elapsed_time );

    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event
    hipEventRecord(start, 0); 
    poly_div4<<<grid, block>>>(d_polinomy, nElem);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // calculate elapsed time
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Measured time for parallel execution = %.6fms\n",
           elapsed_time );
     hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event
    hipEventRecord(start, 0); 
    poly_div5<<<grid, block>>>(d_polinomy, nElem);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // calculate elapsed time
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Measured time for parallel execution = %.6fms\n",
           elapsed_time );

    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event
    hipEventRecord(start, 0); 
    poly_div6<<<grid, block>>>(d_polinomy, nElem);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // calculate elapsed time
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Measured time for parallel execution = %.6fms\n",
           elapsed_time );
    
hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event
    hipEventRecord(start, 0); 
    poly_div7<<<grid, block>>>(d_polinomy, nElem);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // calculate elapsed time
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Measured time for parallel execution = %.6fms\n",
           elapsed_time );

    hipEventCreate(&start);
    hipEventCreate(&stop);
    // record start event
    hipEventRecord(start, 0); 
    poly_div8<<<grid, block>>>(d_polinomy, nElem);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // calculate elapsed time
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Measured time for parallel execution = %.6fms\n",
           elapsed_time );
     
    

    hipMemcpy(h_polinomy, d_polinomy, nBytes, hipMemcpyDeviceToHost);

    hipFree(d_polinomy);
    free(h_polinomy);
}
